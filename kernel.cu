
#include <hip/hip_runtime.h>
#include <math.h>

#define MIN_RGB_VALUE 0
#define MAX_RGB_VALUE 255

#define     TILE_WIDTH      32
#define     SOBEL_WIDTH     3
#define     W           	(TILE_WIDTH + SOBEL_WIDTH)

extern "C" __global__ void shared_sobel_filter(const float * pixin, float * pixout, const int width, const int height)
{	
	__shared__ float cacheImg[W][W];
	
	// Index in actual image
	int idx = (threadIdx.x) + blockDim.x * blockIdx.x;
	
	float sobelMatrix[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
	
	// Destination for pixels in shared memory
    int dest = threadIdx.y * blockDim.y + threadIdx.x;
    int destY = dest / W;
    int destX = dest % W;
    
    // Insert pixels from input image into shared memory
	if(idx < width * height)
        cacheImg[destY][destX] = pixin[idx * 3];
    
    __syncthreads();

    // Perform Sobel convolution
    float px_x = 0;
    float px_y = 0;
    
    for(int y = 0; y < SOBEL_WIDTH; ++y) {
        for(int x = 0; x < SOBEL_WIDTH; ++x) {
			// Magnitude for X
            px_x += cacheImg[threadIdx.y][threadIdx.x + y] * sobelMatrix[x + (y * SOBEL_WIDTH)];
            // Magnitude for Y
            px_y += cacheImg[threadIdx.y][threadIdx.x + y] * sobelMatrix[SOBEL_WIDTH - 1 - y + (x * SOBEL_WIDTH)];
		}
	}
	
	// Compute the gradient magnitude
	float px = (float)(sqrt(px_x * px_x + px_y * px_y));

	// Edge cases of MIN or MAX RGB after the Sobel operator is applied
	if (px < MIN_RGB_VALUE)
		px = MIN_RGB_VALUE;
	else if (px > MAX_RGB_VALUE)
		px = MAX_RGB_VALUE;
	
	// Set the pixel value into the edge image (RGB matrix)
	if(idx < width * height) {
		pixout[idx * 3 + 0] = px;
		pixout[idx * 3 + 1] = px;
		pixout[idx * 3 + 2] = px;
	}
	
    __syncthreads();
}

extern "C" __global__ void sobel_filter(const float * pixin, float * pixout, const int width, const int height)
{	
	int idx = (threadIdx.x) + blockDim.x * blockIdx.x;
    int idy = (threadIdx.y) + blockDim.y * blockIdx.y;
	
	if(idx < width * height) {
		
		// To detect horizontal lines. This is effectively the dy.
		const int sobelX[3][3] = { {-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1} };
		// To detect vertical lines. This is effectively the dy.
		const int sobelY[3][3] = { {-1, -2, -1}, {0, 0, 0}, {1, 2, 1} };

		float px_x = 0;
		float px_y = 0;

		for (int j = 0; j < 3; ++j) {
			for (int i = 0; i < 3; ++i) {
				
				// Index in rows
				const int x = i + idx * 3;
				// Index in colomns
				const int y = j + idy * 3;
				
				const int index = x + y;
				// Magnitude for X
				px_x += pixin[index] * sobelX[i][j];
				// Magnitude for Y
				px_y += pixin[index] * sobelY[i][j];
			}
		}
		
		// Compute the gradient magnitude
		float px = (float)(sqrt(px_x * px_x + px_y * px_y));
		
		// Edge cases of MIN or MAX RGB after the Sobel operator is applied
		if (px < MIN_RGB_VALUE)
			px = MIN_RGB_VALUE;
		else if (px > MAX_RGB_VALUE)
			px = MAX_RGB_VALUE;
		
		// Set the pixel value into the edge image (RGB matrix)
		pixout[idx * 3 + 0] = px;
		pixout[idx * 3 + 1] = px;
		pixout[idx * 3 + 2] = px;
	}
}
    
extern "C" __global__ void gray_scale(const float * pixin, float * pixout, const int width, const int height)
{
    int idx = (threadIdx.x) + blockDim.x * blockIdx.x;
 
	if(idx < width * height) {
		
		// Compute pixels to obtain a grayscale image
		float px = 0.2126 * pixin[idx * 3 + 0] + 
				   0.7152 * pixin[idx * 3 + 1] + 
				   0.0722 * pixin[idx * 3 + 2];
		
		// Save pixel into the output image (RGB matrix)
		pixout[idx * 3 + 0] = px;
		pixout[idx * 3 + 1] = px;
		pixout[idx * 3 + 2] = px;
	}
}

int main(void) {
	return 0;
}